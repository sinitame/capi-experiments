#include "hip/hip_runtime.h"
#include <kernel.h>
#include <stdint.h>


__global__ void add_uint8(uint32_t *A, uint32_t *B, uint32_t *C, int N){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if (id<N){
		C[id] = A[id] + B[id];
	}
}


void cuda_add(uint32_t *A, uint32_t *B, uint32_t *C, int N){

	size_t size = N*sizeof(uint32_t);
	uint32_t *d_A, *d_B, *d_C;

	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	int blockSize = 64;
	int numBlocks = N/64 + 1;

	add_uint8<<<numBlocks,blockSize>>>(d_A,d_B,d_C,N);

	hipMemcpy(C,d_C, size, hipMemcpyDeviceToHost);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

