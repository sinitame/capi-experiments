#include "hip/hip_runtime.h"
#include <kernel.h>

hipStream_t streams[MAX_STREAMS];

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

// Data initialization kernel
__global__ void init_data(uint32_t *buff, const int vector_size){
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	int my_idx = idx;
	while (my_idx < vector_size){
		buff[my_idx] = my_idx;
		my_idx += gridDim.x*blockDim.x; // grid-striding loop
	}
}

// Vector addition kernel
__global__ void vector_add(uint32_t *ibuff, uint32_t *obuff, const int vector_size){
	
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	int my_idx = idx;
	while (my_idx < vector_size){
		obuff[my_idx] = ibuff[my_idx] + ibuff[my_idx];
		my_idx += gridDim.x*blockDim.x; // grid-striding loop
	}
}

void memory_allocation_gpu(uint32_t *buffer[MAX_STREAMS], size_t size){
	int result=0, device_id=0;

	printf("Memory allocation GPU\n");
	hipDeviceGetAttribute (&result, hipDeviceAttributeConcurrentManagedAccess, device_id);
	for (int stream = 0; stream < MAX_STREAMS; stream++){
		checkCuda(hipMallocManaged(&buffer[stream],size));
		if (result) {
			checkCuda(hipMemAdvise(buffer[stream],size,hipMemAdviseSetPreferredLocation,device_id));
		}
		checkCuda(hipMemset(buffer[stream], 0, size));
	}
}

void memory_allocation_host(uint32_t *buffer[MAX_STREAMS], size_t size){

	printf("Memory allocation HOST\n");
	for (int stream = 0; stream < MAX_STREAMS; stream++){
		checkCuda(hipHostAlloc(&buffer[stream], size, hipHostMallocDefault));
	}
}

void init_buffer(uint32_t *buffer[MAX_STREAMS], int vector_size){
	int numBlocks, numThreadsPerBlock = 1024;
	hipDeviceGetAttribute(&numBlocks, hipDeviceAttributeMultiprocessorCount, 0);	
	for (int stream = 0; stream < MAX_STREAMS; stream++){
		init_data<<<4*numBlocks, numThreadsPerBlock>>>(buffer[stream],vector_size);
	}
	hipDeviceSynchronize();

}

void init_streams(){
	for (int stream = 0; stream < MAX_STREAMS; stream++){
		hipStreamCreate(&streams[stream]);
	}
}

void run_new_stream_v1(uint32_t *bufferA, uint32_t *bufferB, uint32_t *ibuff, uint32_t *obuff, int vector_size, int stream){
	int numBlocks, numThreadsPerBlock = 1024;
	size_t size = vector_size*sizeof(uint32_t);
	
	//printf("Running kernel on GPU ..\n");
	hipStream_t stream_i = streams[stream];
	hipDeviceGetAttribute(&numBlocks, hipDeviceAttributeMultiprocessorCount, 0);	
	
	hipMemcpyAsync(ibuff,bufferA, size, hipMemcpyDeviceToHost, stream_i);
	vector_add<<<4*numBlocks, numThreadsPerBlock,0,stream_i>>>(ibuff,obuff,vector_size);
	hipMemcpyAsync(bufferB, obuff, size, hipMemcpyHostToDevice, stream_i);
	hipStreamSynchronize(stream_i);
}

void run_new_stream_v2(uint32_t *ibuff, uint32_t *obuff, int vector_size, int stream){
	int numBlocks, numThreadsPerBlock = 1024;
	hipStream_t stream_i = streams[stream];
	
	hipDeviceGetAttribute(&numBlocks, hipDeviceAttributeMultiprocessorCount, 0);	
	vector_add<<<4*numBlocks, numThreadsPerBlock, 0, stream_i>>>(ibuff,obuff,vector_size);
	hipStreamSynchronize(stream_i);
}


void free_host(uint32_t *buffer[MAX_STREAMS]){
	for (int i = 0; i < MAX_STREAMS; i++){
		hipHostFree(buffer[i]);
	}
}

void free_device(uint32_t *buffer[MAX_STREAMS]){
	for (int i = 0; i < MAX_STREAMS; i++){
		hipFree(buffer[i]);
	}
}
